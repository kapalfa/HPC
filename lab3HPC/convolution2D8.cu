#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.5 
 
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
  int filterL = (2*filterR+1);
  int padding = floor(filterL/2);
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + padding + k;

        sum += h_Src[(y+padding) * imageW + d] * h_Filter[filterR - k];
     
        h_Dst[(y+padding) * imageW + x + padding] = sum;
      }
    }
  }
        
}
////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  int filterL = (2*filterR+1);
  int padding = floor(filterL/2);
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + padding + k;
        sum += h_Src[d * imageW + x + padding] * h_Filter[filterR - k];
 
        h_Dst[(y+padding)* imageW + x+padding] = sum;
      }
    }
  }
}

__global__ void convolution_kernel_x(float *d_output, float *d_input, float *d_filter, int num_row, int num_col, int filterR){

  int idx_x = blockIdx.x*blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y*blockDim.y + threadIdx.y;
  float result = 0.f;
  int filterL = (2*filterR+1);
  int padding = floor(filterL/2);

  for(int k = -filterR; k <= filterR; k++){
    int d = idx_x + padding + k;

    result += d_input[(idx_y+padding)*num_col+d] * d_filter[filterR-k];
  }
  d_output[(idx_y+padding)*num_row+idx_x+padding] = result;
}


__global__ void convolution_kernel_y(float *d_output, float *d_input, float *d_filter, int num_row, int num_col, int filterR){
  int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
  int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
  float result = 0.f;
  int filterL = (2*filterR+1);
  int padding = floor(filterL/2);

  for(int k = -filterR; k<=filterR; k++){
    int d = idx_x + padding + k;

    result += d_input[num_row*d+idx_y+padding] * d_filter[filterR-k];
  }
   
  d_output[(idx_x+padding)*num_row+idx_y+padding] = result;
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_outputGPU;

    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputCPU;

    int imageW;
    int imageH;
    int padding;
    unsigned int i;
    float ap;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    clock_t start_CPU, end_CPU;
    float cpu_time_used;

	  printf("Enter filter radius : ");
	  scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    padding = floor(FILTER_LENGTH/2);

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");

    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc((imageW+2*padding) * (imageH+2*padding) * sizeof(float));
    h_Buffer    = (float *)malloc((imageW+2*padding) * (imageH+2*padding) * sizeof(float));
    h_OutputCPU = (float *)malloc((imageW+2*padding) * (imageH+2*padding) * sizeof(float));
    h_outputGPU = (float *)malloc((imageW+2*padding) * (imageH+2*padding)*sizeof(float));
    if(h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_outputGPU == NULL){
      fprintf(stderr,"Malloc failure: %d\n", __LINE__);
      if (abort) exit(1);
    }

    gpuErrchk(hipMalloc((void**)&d_Filter, FILTER_LENGTH*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_Input, (imageW+2*padding)*(imageH+2*padding)*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_Buffer, (imageW+2*padding)*(imageH+2*padding)*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_OutputCPU, ((imageW+2*padding) * (imageH+2*padding)*sizeof(float))));


    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < (imageW+2*padding) * (imageH+2*padding); i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }

    dim3 grid, block;
    if(imageH > 32){
        block.x = 32; 
        block.y = 32;
        int grid_size = (imageH *imageW)/1024;
        grid.x = sqrt(grid_size);
        grid.y = sqrt(grid_size); 
    }     
    else{
        block.x = imageH;
        block.y = imageH; 
        grid.x = 1;
        grid.y = 1; 
    } 

    hipEventRecord(start);
    gpuErrchk(hipMemcpy(d_Filter, h_Filter, sizeof(float) * FILTER_LENGTH, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_Input, h_Input, (imageW+2*padding) * (imageH+2*padding) * sizeof(float), hipMemcpyHostToDevice));


    convolution_kernel_x<<<grid, block>>>(d_Buffer, d_Input, d_Filter, imageH, imageH, filter_radius);
    hipDeviceSynchronize();

    convolution_kernel_y<<<grid, block>>>(d_OutputCPU, d_Buffer, d_Filter, imageH, imageH, filter_radius);    
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(h_outputGPU, d_OutputCPU, sizeof(float)*(imageW+2*padding) * (imageH+2*padding), hipMemcpyDeviceToHost));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\n\nGPU elapsed time: %f\n\n", milliseconds/1000);

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");


    start_CPU = clock();
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    end_CPU = clock();
    cpu_time_used = ((float) (end_CPU - start_CPU)) / CLOCKS_PER_SEC;
    printf("\n\nCPU elapsed time: %f\n\n", cpu_time_used);

    for(int k = 0; k < (imageW+2*padding) * (imageH+2*padding); k++){
      ap = abs(h_outputGPU[k] - h_OutputCPU[k]);
        if(ap > accuracy){
          printf("OUT OF ACCURACY LEAVE\n");
          return(0);
        }
    }


    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  

    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputCPU);
    hipFree(d_Filter);


    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    hipDeviceReset();


    return 0;
}