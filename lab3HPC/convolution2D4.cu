#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy      0.5
 
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}
////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
}

__global__ void convolution_kernel_x(float *d_output, float *d_input, float *d_filter, int num_row, int num_col, int filterR){

  int idx_x = blockIdx.x*blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y*blockDim.y + threadIdx.y;

  float result = 0.f;
 
  for(int k = -filterR; k <= filterR; k++){
    int d = idx_x + k;

    if(d >= 0 && d < num_row)
      result += d_input[idx_y*num_col+d] * d_filter[filterR-k];
  }
  d_output[idx_y*num_row+idx_x] = result;
}


__global__ void convolution_kernel_y(float *d_output, float *d_input, float *d_filter, int num_row, int num_col, int filterR){
  int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
  int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
  float result = 0.f;

  for(int k = -filterR; k<=filterR; k++){
    int d = idx_x + k;

    if(d >= 0 && d < num_col)
      result += d_input[num_row*d+idx_y] * d_filter[filterR-k];
  }
  d_output[idx_x*num_row+idx_y] = result;
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_outputGPU;

    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputCPU;

    int imageW;
    int imageH;
    unsigned int i;
    float ap;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    clock_t start_CPU, end_CPU;
    float cpu_time_used;

	  printf("Enter filter radius : ");
	  scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");

    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_outputGPU = (float *)malloc(imageW*imageH*sizeof(float));
    if(h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_outputGPU == NULL){
      fprintf(stderr,"Malloc failure: %d\n", __LINE__);
      if (abort) exit(1);
    }


    gpuErrchk(hipMalloc((void**)&d_Filter, FILTER_LENGTH*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_Input, imageW*imageH*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_Buffer, imageW*imageH*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_OutputCPU, imageW*imageH*sizeof(float)));


    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }

    dim3 grid, block;
    if(imageH > 32){
        block.x = 32; 
        block.y = 32;
        int grid_size = (imageH *imageW)/1024;
        grid.x = sqrt(grid_size);
        grid.y = sqrt(grid_size); 
    }     
    else{
        block.x = imageH;
        block.y = imageH; 
        grid.x = 1;
        grid.y = 1; 
    } 
    
    hipEventRecord(start);
    gpuErrchk(hipMemcpy(d_Filter, h_Filter, sizeof(float) * FILTER_LENGTH, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice));

    convolution_kernel_x<<<grid, block>>>(d_Buffer, d_Input, d_Filter, imageH, imageH, filter_radius);
    hipDeviceSynchronize();

    convolution_kernel_y<<<grid, block>>>(d_OutputCPU, d_Buffer, d_Filter, imageH, imageH, filter_radius);    
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(h_outputGPU, d_OutputCPU, sizeof(float)*imageW*imageH, hipMemcpyDeviceToHost));


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\n\nGPU elapsed time: %f\n\n", milliseconds/1000);

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    start_CPU = clock();
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    end_CPU = clock();
    cpu_time_used = ((float) (end_CPU - start_CPU)) / CLOCKS_PER_SEC;
    printf("\n\nCPU elapsed time: %f\n\n", cpu_time_used);
    
    for(int k = 0; k < imageH*imageH; k++){
      ap = abs(h_outputGPU[k] - h_OutputCPU[k]);

        if(ap > accuracy){
          printf("ap = %lf\n", ap);
          printf("OUT OF ACCURACY LEAVE\n");
          return(0);
        }
    }
    
   // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  

    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputCPU);
    hipFree(d_Filter);

    hipEventDestroy(start);
    hipEventDestroy(stop);


    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    hipDeviceReset();


    return 0;
}
